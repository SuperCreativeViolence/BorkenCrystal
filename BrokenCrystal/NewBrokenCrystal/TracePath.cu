#include "hip/hip_runtime.h"

#include "TracePath.h"

__global__ void RenderPathCUDebugKernel(ObjectIntersectionCU* output, ObjectCU** object_list, int num_objects, CameraCU* camera, int* mousePos);
__device__ ObjectIntersectionCU IntersectCU(RayCU* ray, ObjectCU** object_list, int* num_objects, float3* debug_buffer, int thread_index);
__global__ void RenderPathCUKernelLoop(float3* output, ObjectCU** object_list, int* num_objects, CameraCU* camera, float3* debug_buffer, int* loop_x, int* loop_y);
__global__ void RenderPathCUKernel(float3* output, ObjectCU** object_list, int* num_objects, CameraCU* camera, float3* debug_buffer);
__device__ float3 TraceRayCU(RayCU* ray, ObjectCU** object_list, int* num_objects, hiprandState* randState, float3* debug_buffer, int thread_index);
__device__ RayCU GetReflectedRayCU(RayCU* ray, float3 position, float3 normal, float3 color, MaterialType type, hiprandState* randState);



inline __host__ __device__ float clampf(float x) { return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; }
inline __host__ __device__ float cufabs(float x)
{
	return x > 0 ? x : -x;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__device__ RayCU GetReflectedRayCU(RayCU* ray, float3 position, float3 normal, float3 color, MaterialType type, hiprandState* randState)
{
	if (type == SPEC)
	{
		float3 reflected = ray->direction - normal * 2 * dot(normal, ray->direction);
		return RayCU(position, reflected);
	}
	else if (type == GLOSS)
	{
		float roughness = 0.5f;
		float3 reflected = ray->direction - normal * 2 * dot(normal, ray->direction);
		reflected = normalize(make_float3(
			reflected.x + (hiprand_uniform(randState) - 0.5) * roughness,			// random generator for cuda?
			reflected.y + (hiprand_uniform(randState) - 0.5) * roughness,			// random generator for cuda?
			reflected.z + (hiprand_uniform(randState) - 0.5) * roughness			// random generator for cuda?
		));

		return RayCU(position, reflected);
	}
	else if (type == DIFF)
	{
		float3 nl = dot(normal, ray->direction) < 0.0f ? normal : normal * -1.0f;
		float r1 = 2.0f * CU_SIMD_PI * hiprand_uniform(randState);						// random generator for cuda?
		float r2 = hiprand_uniform(randState);									// random generator for cuda?
		float r2s = sqrt(r2);

		float3 w = nl;
		float3 u;
		if (cufabs(w.x) > 0.1)
			u = normalize(cross(make_float3(0.0f, 1.0f, 0.0f), w));
		else
			u = normalize(cross(make_float3(1.0f, 0.0f, 0.0f), w));
		float3 v = cross(w, u);
		float3 d = normalize((u * cos(r1) * r2s + v * sin(r1) * r2s + w * sqrt(1 - r2)));
		return RayCU(position, d);
	}
	else if (type == TRANS)
	{
		float3 nl = dot(normal, ray->direction) < 0 ? normal : normal * -1;
		float3 reflection = ray->direction - normal * 2 * dot(normal, ray->direction);
		bool into = dot(normal, nl) > 0;
		float nc = 1.0f;
		float nt = 1.5f;
		float nnt;

		float Re, RP, TP, Tr;
		float3 tdir = make_float3(0.0f, 0.0f, 0.0f);

		if (into)
			nnt = nc / nt;
		else
			nnt = nt / nc;

		float ddn = dot(ray->direction, nl);
		float cos2t = 1.0f - nnt * nnt * (1.0f - ddn * ddn);

		if (cos2t < 0) return RayCU(position, reflection);

		if (into)
			tdir = normalize((ray->direction * nnt - normal * (ddn * nnt + sqrt(cos2t))));
		else
			tdir = normalize((ray->direction * nnt + normal * (ddn * nnt + sqrt(cos2t))));

		float a = nt - nc;
		float b = nt + nc;
		float R0 = a * a / (b * b);

		float c;
		if (into)
			c = 1 + ddn;
		else
			c = 1 - dot(tdir, normal);

		Re = R0 + (1 - R0) * c * c * c * c * c;
		Tr = 1 - Re;

		float P = .25 + .5 * Re;
		RP = Re / P;
		TP = Tr / (1 - P);

		if (hiprand_uniform(randState) < P)		// random generator for cuda?
		{
			color = color * (RP);
			return RayCU(position, reflection);
		}

		color = color * (TP);
		return RayCU(position, tdir);
	}
	else {}
}


__global__ void initCURand(unsigned int seed, hiprandState_t* states)
{

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		blockIdx.x, /* the sequence number should be different for each core (unless you want all
					cores to get the same sequence of numbers for some reason - use thread id! */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[blockIdx.x]);
}

void TracePath::RenderPathCUDebug(ObjectCU** object_list, int num_objects, CameraCU* camera, int* mousePos)
{
	ObjectIntersectionCU* output_host = new ObjectIntersectionCU;
	ObjectIntersectionCU* output_device;

	int* mousePos_device;

	hipMalloc((void**)&output_device, sizeof(ObjectIntersectionCU));
	hipMalloc((void**)&mousePos_device, sizeof(int) * 2);

	hipMemcpy(mousePos_device, mousePos, sizeof(int) * 2, hipMemcpyHostToDevice);

	dim3 block(1, 1, 1);
	dim3 grid(1, 1, 1);


	RenderPathCUDebugKernel << <grid, block >> > (output_device, object_list, num_objects, camera, mousePos_device);

	hipMemcpy(output_host, output_device, sizeof(ObjectIntersectionCU), hipMemcpyDeviceToHost);

	if (output_host[0].hit == 0)
	{
		std::cout << "No Hit!" << std::endl;
	}
	else if (output_host[0].material == EMIT)
	{
		std::cout << "EMIT!" << std::endl;
	}
	else
	{
		printf("Hit : %f | normal :  %.1f %.1f %.1f | color : %.1f %.1f %.1f\n", output_host[0].u, output_host[0].normal.x, output_host[0].normal.y, output_host[0].normal.z, output_host[0].color.x, output_host[0].color.y, output_host[0].color.z);
	}


	hipFree(output_device);
	hipFree(object_list);
	hipFree(camera);
	hipFree(mousePos_device);
	delete output_host;
}

__global__ void RenderPathCUDebugKernel(ObjectIntersectionCU* output, ObjectCU** object_list, int num_objects, CameraCU* camera, int* mousePos)
{
	hiprandState randState;
	hiprand_init(0, 0, 0, &randState);
	RayCU ray = camera->GetRay(&randState, mousePos[0], mousePos[1], 0, 0, 0);
	ObjectIntersectionCU intersection = ObjectIntersectionCU();
	ObjectIntersectionCU temp = ObjectIntersectionCU();	// return value of objects.at((unsigned)i)->GetIntersection(ray)
	ObjectCU* current_obj;

	ObjectIntersectionCU temp_inner = ObjectIntersectionCU(); // return value of triangle->GetIntersect()

	for (int i = 0; i < num_objects; i++)
	{
		current_obj = object_list[i];

		float tNear = FLT_MAX_CU;

		for (unsigned int j = 0; j < current_obj->triangles_num; j += 3)
		{
			float3 v0 = current_obj->triangles_p[j];
			float3 v1 = current_obj->triangles_p[j + 1];
			float3 v2 = current_obj->triangles_p[j + 2];

			// triangle->GetIntersection(ray, transform)

			int hit = 0;
			float u, v, t = 0;

			float3 normal = normalize(cross(v1 - v0, v2 - v0));

			float3 v0v1 = v1 - v0;
			float3 v0v2 = v2 - v0;
			float3 pvec = cross(ray.direction, v0v2);
			float det = dot(v0v1, pvec);

			if (cufabs(det) < EPSILON_CU)
			{

				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			float3 tvec = ray.origin - v0;
			u = dot(tvec, pvec);

			if (u < 0 || u > det)
			{

				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit == 1 && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			float3 qvec = cross(tvec, v0v1);
			v = dot(ray.direction, qvec);

			if (v < 0 || u + v > det)
			{
				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			t = dot(v0v2, qvec) / det;

			if (t < EPSILON_CU)
			{
				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			hit = 1;

			temp_inner.hit = hit;
			temp_inner.material = current_obj->material;
			temp_inner.u = t;
			temp_inner.normal = normal;
			if (temp_inner.hit && temp_inner.u < tNear)
			{
				tNear = temp_inner.u;
				temp.hit = temp_inner.hit;
				temp.material = temp_inner.material;
				temp.normal = temp_inner.normal;
				temp.u = temp_inner.u;
			}
		}

		if (temp.hit == 1)
		{
			if (intersection.u == 0 || temp.u < intersection.u)
			{
				intersection.hit = temp.hit;
				intersection.material = temp.material;
				intersection.normal = temp.normal;
				intersection.u = temp.u;
				intersection.color = current_obj->color;
				intersection.emission = current_obj->emission;
			}
		}
	}
	output->color = intersection.color;
	output->emission = intersection.emission;
	output->hit = intersection.hit;
	output->material = intersection.material;
	output->normal = intersection.normal;
	output->u = intersection.u;
}



float3* TracePath::RenderPathCU(ObjectCU** object_list, int* num_objects, CameraCU* camera, int width, int height) // need to use camera not CameraCU but Camera in Scene
{
	// time
	unsigned int startTime = time(nullptr);

	std::cout << "RenderPathCU entered successfully" << std::endl;
	float3* output_host = new float3[width * height];
	float3* output_device;

	float3* debug_host = new float3[width * height];
	float3* debug_device;


	hipMalloc(&output_device, width * height * sizeof(float3));

	hipMalloc(&debug_device, width * height * sizeof(float3));

	std::cout << "output_device hipMalloc successed" << std::endl;

	dim3 block(16, 9, 1); // calculate this

	size_t blocks_width = ceilf(width / block.x);
	size_t blocks_height = ceilf(height / block.y);

	dim3 grid(20, 20, 1);  // calculate this
	//dim3 grid(blocks_width, blocks_height, 1);

	std::cout << "dim set successed" << std::endl;
	std::cout << "block : " << block.x << block.y << block.z << " grid : " << grid.x << grid.y << grid.z << std::endl;

	// loop mem allocate
	int* loop_x_device;
	int* loop_y_device;
	int loop_x_host = 0;
	int loop_y_host = 0;
	hipMalloc((void**)&loop_x_device, sizeof(int));
	hipMalloc((void**)&loop_y_device, sizeof(int));


	// loop version
	for (loop_x_host = 0; loop_x_host < TRACE_SAMPLES_LOOP_X; loop_x_host++)
	{
		for (loop_y_host = 0; loop_y_host < TRACE_SAMPLES_LOOP_Y; loop_y_host++)
		{
			hipMemcpy(loop_x_device, &loop_x_host, sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(loop_y_device, &loop_y_host, sizeof(int), hipMemcpyHostToDevice);

			RenderPathCUKernelLoop << < grid, block >> > (output_device, object_list, num_objects, camera, debug_device, loop_x_device, loop_y_device);
			
			hipError_t error = hipDeviceSynchronize();
			if (error != hipSuccess)
			{
				// print the CUDA error message and exit
				printf("CUDA error: %s\n", hipGetErrorString(error));
			}
			std::cout << "Current loop: " << loop_x_host << " " << loop_y_host << std::endl;
			//Sleep(1000);
		}
	}

	// no loop version

	//RenderPathCUKernel << < grid, block >> > (output_device, object_list, num_objects, camera, debug_device);
	//Sleep(100);
	/*
	hipError_t error = hipDeviceSynchronize();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}*/



	
	// make the host block until the device is finished with foo

	// check for error
	/*error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}*/

	std::cout << "RenderPathCUKernel successed" << std::endl;

	hipMemcpy(output_host, output_device, width * height * sizeof(float3), hipMemcpyDeviceToHost);

	hipMemcpy(debug_host, debug_device, width * height * sizeof(float3), hipMemcpyDeviceToHost);

	std::cout << "copy result device to host successed" << std::endl;

	/* debug */
	/*for (int i = 0; i < width * height; i++)
	{
		//std::cout << debug_host[i].x << " " << debug_host[i].y << " " << debug_host[i].z << std::endl;
		//if(debug_host[i].x == 4.0f)
		//std::cout << debug_host[i].x <<" "<<debug_host[i].y << " " << debug_host[i].z << std::endl;
	}*/
	/* debug end */
	/*	
	for (int i = 0; i < num_objects; i++)
	{
		hipFree(object_list[i]->triangles_p);
		hipFree(object_list);
	}*/
	
	// check for error

	hipFree(output_device);
	hipFree(object_list);
	hipFree(camera);
	hipFree(num_objects);
	hipFree(loop_x_device);
	hipFree(loop_y_device);
	hipFree(debug_device);
	delete debug_host;

	std::cout << "hipFree successed" << std::endl;

	int elapsedTime = (int)difftime(time(nullptr), startTime);
	printf("\rCUDA PathTracing complete, time taken: %.2dh%.2dm%.2ds.\n", elapsedTime / 3600, (elapsedTime % 3600) / 60, elapsedTime % 60);

	return output_host;
}

__global__ void RenderPathCUKernel(float3* output, ObjectCU** object_list, int* num_objects, CameraCU* camera, float3* debug_buffer)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState randState;
	hiprand_init(threadId, 0, 0, &randState);
	int i = y * camera->width + x;


	if (i < camera->width * camera->height)
	{
		//debug_buffer[i] = make_float3(0.0f, 0.0f, 0.0f);

		float samplesP = 1.0f / TRACE_SAMPLES;
		float3 resultcolor = make_float3(0.0f, 0.0f, 0.0f);

		//debug_buffer[i].x = hiprand_uniform(&randState);
		//debug_buffer[i].y = hiprand_uniform(&randState);

		for (int sy = 0; sy < 2; sy++)
		{
			for (int sx = 0; sx < 2; sx++)
			{
				float3 color = make_float3(0.0f, 0.0f, 0.0f);
				for (int s = 0; s < TRACE_SAMPLES; s++)
				{
					RayCU ray = camera->GetRay(&randState, x, y, sx, sy, 0); // ***
																			 //ray = camera->GetRay(&randState, x, y, sx, sy, 0); // ***
					color = color + TraceRayCU(&ray, object_list, num_objects, &randState, debug_buffer, i);
					//free(&ray);

				}

				resultcolor = resultcolor + (color * samplesP);
			}
		}

		resultcolor = resultcolor * 0.25f;
		output[i] = make_float3(clampf(resultcolor.x), clampf(resultcolor.y), clampf(resultcolor.z));

		free(&randState);
		/* debug */
		//debug_buffer[i].x = i;
		/* debug end */
	}
}

__global__ void RenderPathCUKernelLoop(float3* output, ObjectCU** object_list, int* num_objects, CameraCU* camera, float3* debug_buffer, int* loop_x, int* loop_y)
{
	int x = gridDim.x * blockDim.x * loop_x[0] + blockIdx.x * blockDim.x + threadIdx.x;
	int y = gridDim.y * blockDim.y * loop_y[0] + blockIdx.y * blockDim.y + threadIdx.y;

	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState randState;
	hiprand_init(threadId, 0, 0, &randState);
	int i = y * camera->width + x;


	if (i < camera->width * camera->height)
	{
		//debug_buffer[i] = make_float3(0.0f, 0.0f, 0.0f);
		
		float samplesP = 1.0f / TRACE_SAMPLES;
		float3 resultcolor = make_float3(0.0f, 0.0f, 0.0f);

		//debug_buffer[i].x = hiprand_uniform(&randState);
		//debug_buffer[i].y = hiprand_uniform(&randState);

		for (int sy = 0; sy < 2; sy++)
		{
			for (int sx = 0; sx < 2; sx++)
			{
				float3 color = make_float3(0.0f, 0.0f, 0.0f);
				for (int s = 0; s < TRACE_SAMPLES; s++)
				{
					RayCU ray = camera->GetRay(&randState, x, y, sx, sy, 0); // ***
					//ray = camera->GetRay(&randState, x, y, sx, sy, 0); // ***
					color = color + TraceRayCU(&ray, object_list, num_objects, &randState, debug_buffer, i);
					//free(&ray);

				}

				resultcolor = resultcolor + (color * samplesP);
			}
		}

		resultcolor = resultcolor * 0.25f;
		output[i] = make_float3(clampf(resultcolor.x), clampf(resultcolor.y), clampf(resultcolor.z));
		
		free(&randState);
		/* debug */
		//debug_buffer[i].x = i;
		/* debug end */
	}
}

__device__ float3 TraceRayCU(RayCU* ray, ObjectCU** object_list, int* num_objects, hiprandState* randState, float3* debug_buffer, int thread_index)
{
	float3 result_color = make_float3(.0f, .0f, .0f);
	
	for (int depth = 0; depth < 15; depth++)
	{
		ObjectIntersectionCU intersection = IntersectCU(ray, object_list, num_objects, debug_buffer, thread_index);
		// debug 
		//debug_buffer[thread_index].x = (int)intersection.material;

		// debug end
		
		if (intersection.hit == 0) return make_float3(.0f, .0f, .0f);
		if (intersection.material == EMIT)
		{
			//debug_buffer[thread_index].x += 1.0f;
			if (depth == 0)
			{
				return intersection.emission;
			}
			else
			{
				return result_color * intersection.emission;
			}
		}	// need to be fixed

		float3 color = intersection.color;
		float maxReflection = color.x > color.y && color.x > color.z ? color.x : color.y > color.z ? color.y : color.z;
		float random = hiprand_uniform(randState);// random number generator for cuda?

		if (depth > 5)
		{
			if (random < maxReflection * 0.9f)
			{
				color = color * (0.9f / maxReflection);
			}
			else
			{
				return result_color * intersection.emission;
			}
		}

		if (depth == 0)
		{
			result_color = color;
		}
		else
		{
			result_color = result_color * color;
		}


		float3 pos = ray->origin + ray->direction * intersection.u;
		RayCU reflected = GetReflectedRayCU(ray, pos, intersection.normal, color, intersection.material, randState);
		ray = &reflected;
	}
	return result_color;

}


__device__ ObjectIntersectionCU IntersectCU(RayCU* ray, ObjectCU** object_list, int* num_objects, float3* debug_buffer, int thread_index)
{
	ObjectIntersectionCU intersection = ObjectIntersectionCU();
	ObjectIntersectionCU temp = ObjectIntersectionCU();	// return value of objects.at((unsigned)i)->GetIntersection(ray)
	ObjectCU* current_obj;

	ObjectIntersectionCU temp_inner = ObjectIntersectionCU(); // return value of triangle->GetIntersect()

	for (int i = 0; i < num_objects[0]; i++)
	{
		current_obj = object_list[i];

		float tNear = FLT_MAX_CU;

		//debug_buffer[thread_index].z = num_objects;
		for (unsigned int j = 0; j < current_obj->triangles_num; j += 3)
		{
		
			//debug_buffer[thread_index].z += 1.0f;


			float3 v0 = current_obj->triangles_p[j];
			float3 v1 = current_obj->triangles_p[j + 1];
			float3 v2 = current_obj->triangles_p[j + 2];
			
			// triangle->GetIntersection(ray, transform)

			int hit = 0;
			float u, v, t = 0;

			float3 normal = normalize(cross(v1 - v0, v2 - v0));

			float3 v0v1 = v1 - v0;
			float3 v0v2 = v2 - v0;
			float3 pvec = cross(ray->direction, v0v2);
			float det = dot(v0v1, pvec);

			if (cufabs(det) < EPSILON_CU)
			{

				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			float3 tvec = ray->origin - v0;
			u = dot(tvec, pvec);

			if (u < 0 || u > det)
			{

				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit == 1 && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			float3 qvec = cross(tvec, v0v1);
			v = dot(ray->direction, qvec);

			if (v < 0 || u + v > det)
			{

				//debug_buffer[thread_index].z += 1.0f;
				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			t = dot(v0v2, qvec) / det;

			if (t < EPSILON_CU)
			{

				//debug_buffer[thread_index].x += 1.0f;
				temp_inner.hit = hit;
				temp_inner.material = current_obj->material;
				temp_inner.u = t;
				temp_inner.normal = normal;
				if (temp_inner.hit && temp_inner.u < tNear)
				{
					tNear = temp_inner.u;
					temp.hit = temp_inner.hit;
					temp.material = temp_inner.material;
					temp.normal = temp_inner.normal;
					temp.u = temp_inner.u;
				}
				continue;
			}

			hit = 1;

			//debug_buffer[thread_index].y += 1.0f;

			temp_inner.hit = hit;
			temp_inner.material = current_obj->material;
			temp_inner.u = t;
			temp_inner.normal = normal;
			if (temp_inner.hit && temp_inner.u < tNear)
			{
				tNear = temp_inner.u;
				temp.hit = temp_inner.hit;
				temp.material = temp_inner.material;
				temp.normal = temp_inner.normal;
				temp.u = temp_inner.u;
			}
		}
		
		if (temp.hit == 1)
		{

			//if (temp.material == EMIT) debug_buffer[thread_index].x += 1.0f;
			if (intersection.u == 0 || temp.u < intersection.u)
			{
				//debug_buffer[thread_index].y += 1.0f;
				intersection.hit = temp.hit;
				intersection.material = temp.material;
				intersection.normal = temp.normal;
				intersection.u = temp.u;
				intersection.color = current_obj->color;
				intersection.emission = current_obj->emission;
			}
		}
	}
	/*
	free(&temp);
	free(current_obj);
	free(&temp_inner);*/

	return intersection;
}
